#include "hip/hip_runtime.h"
%%cu
#include<iostream>
#include<algorithm>
#include<vector>
#include<cassert>
const int outwidth = 4;

__global__ void temp(int * input, int *output, int * mask, int width)
{
    int tx = threadIdx.x;
    int index_o = blockIdx.x * outwidth + threadIdx.x;
    int index_i = index_o -2;
    int temp1 = 0;
    __shared__ int input_s[8];
    if ((index_i) >= 0 && (index_i) < width)
    {
        input_s[tx] = input[index_i];
    }
    else
    {
        input_s[tx] = 0;
    }
    __syncthreads();
    if (tx < outwidth)
    {
        temp1 = 0;
        for (int j = 0; j < 5; j++)
            temp1 += mask[j] * input_s[j + tx];
        output[index_o] = temp1;
    }
}

void serial_temp(const std::vector<int>& input, const std::vector<int>& mask, std::vector<int>& output, int width)
{
    for (int i = 0; i < width; i++)
    {
        int temp = 0;
        for (int j = 0; j < 5; j++)
        {
            int index = i - (5 / 2) + j;
            if (index >= 0 && index < width)
            {
                temp += mask[j] * input[index];
            }
        }
        output[i] = temp;
    }
}
int main()
{
    int width = 1024;
    size_t bytes = sizeof(int) * width;
    // Host data
    std::vector<int> input;
    input.reserve(width);
    std::vector<int> mask;
    mask.reserve(5);
    std::vector<int> output;
    output.reserve(width);
    // Generate random input and mask
    for (int i = 0; i < width; i++)
    {
        input.push_back(rand() % 100);
    }
    for (int i = 0; i < 5; i++)
    {
        mask.push_back(rand() % 100);
    }
    // Allocate device memory
    int* d_i, *d_m, *d_o;
    hipMalloc(&d_i, bytes);
    hipMalloc(&d_m, 5 * sizeof(int));
    hipMalloc(&d_o, bytes);

    hipMemcpy(d_i, input.data(), bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_m, mask.data(), 5 * sizeof(int), hipMemcpyHostToDevice);
    // Launch GPU kernel
    int threads = 99;
    int blocks = (width + threads - 1) / threads;
    temp<<<blocks, threads>>>(d_i, d_o, d_m, width);
    hipMemcpy(output.data(), d_o, bytes, hipMemcpyDeviceToHost);
    std::vector<int> serial_output(width);
    serial_temp(input, mask, serial_output, width);
    for (int i = 0; i < 5; i++)
    {
        std::cout << "Element " << i << " - GPU: " << output[i] << " | Serial: " << serial_output[i] << std::endl;
    }
    std::cout << "Verification completed." << std::endl;
    hipFree(d_i);
    hipFree(d_m);
    hipFree(d_o);
    return 0;
}






