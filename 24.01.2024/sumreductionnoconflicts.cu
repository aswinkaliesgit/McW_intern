#include "hip/hip_runtime.h"
%%cuda


#include <hip/hip_runtime.h>
#include <>
#include <stdlib.h>
#include <stdio.h>
#include <assert.h>
#include <math.h>

#define SIZE 256
#define SHMEM_SIZE 256 * 4

__global__ void sum_reduction(int *v, int *v_r) {
	
	__shared__ int partial_sum[SHMEM_SIZE];


	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	partial_sum[threadIdx.x] = v[tid];
	__syncthreads();

	for (int s = blockDim.x / 2; s > 0; s >>= 1) {
		if (threadIdx.x < s) {
			partial_sum[threadIdx.x] += partial_sum[threadIdx.x + s];
		}
		__syncthreads();
	}

	
	if (threadIdx.x == 0) {
		v_r[blockIdx.x] = partial_sum[0];
	}
}

void initialize_vector(int *v, int n) {
	for (int i = 0; i < n; i++) {
		v[i] = 1;//rand() % 10;
	}
}

int main() {

	int n = 1 << 16;
	size_t bytes = n * sizeof(int);

	
	int *h_v, *h_v_r;
	int *d_v, *d_v_r;

	h_v = (int*)malloc(bytes);
	h_v_r = (int*)malloc(bytes);
	hipMalloc(&d_v, bytes);
	hipMalloc(&d_v_r, bytes);

	initialize_vector(h_v, n);

	hipMemcpy(d_v, h_v, bytes, hipMemcpyHostToDevice);

	int TB_SIZE = SIZE;

	int GRID_SIZE = n / TB_SIZE;


	sum_reduction <<<GRID_SIZE, TB_SIZE >>> (d_v, d_v_r);

	sum_reduction <<<1, TB_SIZE >>> (d_v_r, d_v_r);

	hipMemcpy(h_v_r, d_v_r, bytes, hipMemcpyDeviceToHost);

	assert(h_v_r[0] == 65536);

	printf("COMPLETED SUCCESSFULLY\n");

	return 0;
}