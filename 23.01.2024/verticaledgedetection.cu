#include "hip/hip_runtime.h"
%%cu
#include<iostream>
#include<cassert>
#include<cstdlib>
#include <fstream>
#include <cstdint>
#include <vector>

#define maskdim 3

#define maskoffset (maskdim/2)
#define N 575
__constant__ int mask[maskdim*maskdim];
#pragma pack(push, 1)

struct BitmapHeader {
    uint16_t signature;
    uint32_t fileSize;
    uint32_t reserved;
    uint32_t dataOffset;
    uint32_t headerSize;
    int32_t  width;
    int32_t  height;
    uint16_t planes;
    uint16_t bitDepth;
    uint32_t compression;
    uint32_t imageSize;
    int32_t  xPixelsPerMeter;
    int32_t  yPixelsPerMeter;
    uint32_t colorsUsed;
    uint32_t colorsImportant;
};
#pragma pack(pop)

struct Pixel {
    uint8_t blue;
    uint8_t green;
    uint8_t red;
};

struct Bitmap {
    BitmapHeader header;
    std::vector<Pixel> pixels;
};

void readBitmap(const char* filename, Bitmap& bitmap) {
    std::ifstream file(filename, std::ios::binary);

    if (!file.is_open()) {
        std::cerr << "Error opening file: " << filename << std::endl;
        return;
    }

    file.read(reinterpret_cast<char*>(&bitmap.header), sizeof(BitmapHeader));

    if (bitmap.header.signature != 0x4D42 || bitmap.header.bitDepth != 24 || bitmap.header.compression != 0) {
        std::cerr << "Invalid BMP file format or unsupported features." << std::endl;
        file.close();
        return;
    }

    uint32_t rowSize = (bitmap.header.width * 3 + 3) & ~3;
    bitmap.pixels.resize(bitmap.header.height * bitmap.header.width);

    for (int y = bitmap.header.height - 1; y >= 0; --y) {
        file.read(reinterpret_cast<char*>(bitmap.pixels.data() + y * bitmap.header.width), rowSize);
    }

    file.close();
}

void writeBitmap(const char* filename, const Bitmap& bitmap) {
    std::ofstream file(filename, std::ios::binary);

    if (!file.is_open()) {
        std::cerr << "Error opening file for writing: " << filename << std::endl;
        return;
    }

    file.write(reinterpret_cast<const char*>(&bitmap.header), sizeof(BitmapHeader));

    uint32_t rowSize = (bitmap.header.width * 3 + 3) & ~3;

    for (int y = bitmap.header.height - 1; y >= 0; --y) {
        file.write(reinterpret_cast<const char*>(bitmap.pixels.data() + y * bitmap.header.width), rowSize);
    }

    file.close();
}

__global__ void convolution_2d(Pixel *matrix, Pixel *result)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    int startrow = row - maskoffset;
    int startcol = col - maskoffset;
    int temp1 = 0;
    int temp2 = 0;
    int temp3 = 0;

    for (int i = 0; i < maskdim; i++)
    {
        for (int j = 0; j < maskdim; j++)
        {
            if (startrow + i >= 0 && (startrow + i) < N && startcol + j >= 0 && (startcol + j) < N)
            {
                temp1 += matrix[(startrow + i) * N + (startcol + j)].red * mask[i * maskdim + j];
                temp2 += matrix[(startrow + i) * N + (startcol + j)].blue * mask[i * maskdim + j];
                temp3 += matrix[(startrow + i) * N + (startcol + j)].green * mask[i * maskdim + j];
            }
        }
    }

    if (row < N && col < N)
    {
        result[row * N + col].red = temp1;
        result[row * N + col].blue = temp2;
        result[row * N + col].green = temp3;
    }
}

int main()
{
    int *result = new int[N * N];

    int hmask[3 * 3] = {-1, 0, 1, -2, 0, 2, -1, 0, 1};

    hipMemcpyToSymbol(HIP_SYMBOL(mask), hmask, sizeof(int) * maskdim * maskdim);
    const char *filename = "input.bmp";
    Bitmap bitmap;

    readBitmap(filename, bitmap);
    std::cout << "Width: " << bitmap.header.width << std::endl;
    Pixel *d_pixel;
    Pixel *d_result;
    hipMalloc(&d_pixel, bitmap.header.height * bitmap.header.width * sizeof(Pixel));
    hipMalloc(&d_result, bitmap.header.height * bitmap.header.width * sizeof(Pixel));
    hipMemcpy(d_pixel, bitmap.pixels.data(), bitmap.header.height * bitmap.header.width * sizeof(Pixel), hipMemcpyHostToDevice);

    std::cout << "Width: " << bitmap.header.width << std::endl;
    std::cout << "Height: " << bitmap.header.height << std::endl;

    dim3 blockSize(16, 16);
    dim3 gridSize((bitmap.header.width + blockSize.x - 1) / blockSize.x, (bitmap.header.height + blockSize.y - 1) / blockSize.y);
    convolution_2d<<<gridSize, blockSize>>>(d_pixel, d_result);

    std::vector<Pixel> a(bitmap.header.height * bitmap.header.width);
    hipMemcpy(a.data(), d_result, bitmap.header.height * bitmap.header.width * sizeof(Pixel), hipMemcpyDeviceToHost);

    Bitmap modifiedBitmap;
    modifiedBitmap.header = bitmap.header;
    modifiedBitmap.pixels = a;

    writeBitmap("output.bmp", modifiedBitmap);

    hipFree(d_pixel);
    hipFree(d_result);

    delete[] result;

    return 0;
}
