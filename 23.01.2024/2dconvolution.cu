#include "hip/hip_runtime.h"
%%cu
#include<iostream>
#include<cassert>
#include<cstdlib>

#define maskdim 3

#define maskoffset (maskdim/2)
#define N 1024
__constant__ int mask[maskdim*maskdim];

__global__ void convolution_2d(int *matrix, int* result)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    int startrow = row - maskoffset;
    int startcol = col - maskoffset;
    int temp = 0;

    for (int i = 0; i < maskdim; i++)
    {
        for (int j = 0; j < maskdim; j++)
        {
            if (startrow+i >= 0 && (startrow + i) < N && startcol+j >= 0 && (startcol + j) < N)
            {
                temp += matrix[(startrow + i) * N + (startcol + j)] * mask[i * maskdim + j];
            }
        }
    }

    if (row < N && col < N) {
        result[row * N + col] = temp;
    }
}

void verify_result(int *m, int *mask, int *result) {
    int temp;
    int offset_r;
    int offset_c;

    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            temp = 0;

            for (int k = 0; k < maskdim; k++) {
                offset_r = i - maskoffset + k;

                for (int l = 0; l < maskdim; l++) {
                    offset_c = j - maskoffset + l;

                    if (offset_r >= 0 && offset_r < N && offset_c >= 0 && offset_c < N) {
                        temp += m[offset_r * N + offset_c] * mask[k * maskdim + l];
                    }
                }
            }
         //std::cout<<result[i * N + j]<<"\t"<<temp<<"\n";
            assert(result[i * N + j] == temp);
        }
    }
}

int main()
{
    size_t bytes = sizeof(int) * N * N;
    int *matrix = new int[N * N];
    int *result = new int[N * N];
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            matrix[N * i + j] = rand() % 100;
        }
    }

    int *hmask = new int[maskdim * maskdim];
    for (int i = 0; i < maskdim; i++) {
        for (int j = 0; j < maskdim; j++) {
            hmask[maskdim * i + j] = rand() % 100;
        }
    }

    hipMemcpyToSymbol(HIP_SYMBOL(mask), hmask, sizeof(int) * maskdim * maskdim);

    int *d_matrix;
    int *d_result;
    hipMalloc(&d_matrix, bytes);
    hipMalloc(&d_result, bytes);
    hipMemcpy(d_matrix, matrix, bytes, hipMemcpyHostToDevice);

    int THREADS = 16;
    int BLOCKS = (N + THREADS - 1) / THREADS;
    dim3 block_dim(THREADS, THREADS);
    dim3 grid_dim(BLOCKS, BLOCKS);

    convolution_2d<<<grid_dim, block_dim>>>(d_matrix, d_result);
    hipMemcpy(result, d_result, bytes, hipMemcpyDeviceToHost);

    verify_result(matrix, hmask, result);

    std::cout << "COMPLETED SUCCESSFULLY!\n";
    delete[] matrix;
    delete[] result;
    delete[] hmask;

    hipFree(d_matrix);
    hipFree(d_result);

    return 0;
}
