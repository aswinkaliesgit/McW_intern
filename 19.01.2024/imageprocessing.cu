#include "hip/hip_runtime.h"
%%cu
#include <iostream>
#include <fstream>
#include <cstdint>
#include <vector>

#pragma pack(push, 1) // Pack structures without padding

struct BitmapHeader {
    uint16_t signature;     
    uint32_t fileSize;      
    uint32_t reserved;     
    uint32_t dataOffset;    
    uint32_t headerSize;    
    int32_t  width;         
      int32_t  height;        
      uint16_t planes;        
    uint16_t bitDepth;    
    uint32_t compression;   
    uint32_t imageSize;    
    int32_t  xPixelsPerMeter; 
    int32_t  yPixelsPerMeter; 
    uint32_t colorsUsed;    
    uint32_t colorsImportant; 
};
#pragma pack(pop)

struct Pixel {
    uint8_t blue;
    uint8_t green;
    uint8_t red;
};

struct Bitmap {
    BitmapHeader header;
    std::vector<Pixel> pixels;
};

void readBitmap(const char* filename, Bitmap& bitmap) {
    std::ifstream file(filename, std::ios::binary);

    if (!file.is_open()) {
        std::cerr << "Error opening file: " << filename << std::endl;
        return;
    }

    file.read(reinterpret_cast<char*>(&bitmap.header), sizeof(BitmapHeader));

    if (bitmap.header.signature != 0x4D42 || bitmap.header.bitDepth != 24 || bitmap.header.compression != 0) {
        std::cerr << "Invalid BMP file format or unsupported features." << std::endl;
        file.close();
        return;
    }

    uint32_t rowSize = (bitmap.header.width * 3 + 3) & ~3; 
    bitmap.pixels.resize(bitmap.header.height * bitmap.header.width);

    for (int y = bitmap.header.height - 1; y >= 0; --y) {
        file.read(reinterpret_cast<char*>(bitmap.pixels.data() + y * bitmap.header.width), rowSize);
    }

    file.close();
}
__global__  void temp(Pixel *x,int size)
{
    int id= blockDim.x*blockIdx.x + threadIdx.x;
    if(id<size)
    {
        x[id].red=255-x[id].red;
          x[id].blue=255-x[id].blue;
          x[id].green=255-x[id].green;
    }
}

int main() {
    const char* filename = "input.bmp";
    Bitmap bitmap;

    readBitmap(filename, bitmap);
    std::cout << "Width: " << bitmap.header.width << std::endl;
    std::cout << "Height: " << bitmap.header.height << std::endl;
    Pixel firstPixel = bitmap.pixels[0];
    std::cout << "First Pixel - R: " << static_cast<int>(bitmap.pixels[10].red) << " G: " << static_cast<int>(firstPixel.green) << " B: " << static_cast<int>(firstPixel.blue) << std::endl;

    Pixel *d_pixel ;
    hipMalloc(&d_pixel, bitmap.header.height * bitmap.header.width * sizeof(Pixel));
   hipMemcpy(d_pixel, bitmap.pixels.data(), bitmap.header.height * bitmap.header.width * sizeof(Pixel), hipMemcpyHostToDevice);
    int threadsize=1024;
    int blocks=(bitmap.header.height * bitmap.header.width+threadsize - 1) / threadsize;

  temp<<<blocks,threadsize>>>(d_pixel,bitmap.header.height * bitmap.header.width);
 std::vector<Pixel> a;
  a.reserve(bitmap.header.height * bitmap.header.width);


     hipMemcpy(a.data(), d_pixel, bitmap.header.height * bitmap.header.width * sizeof(Pixel), hipMemcpyDeviceToHost);
     std::cout<<(int(a[10].red));
  return 0;
}


