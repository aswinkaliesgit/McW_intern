#include "hip/hip_runtime.h"
%%cu
#include <iostream>
#include <fstream>
#include <cstdint>
#include <vector>

#pragma pack(push, 1)

struct BitmapHeader {
    uint16_t signature;
    uint32_t fileSize;
    uint32_t reserved;
    uint32_t dataOffset;
    uint32_t headerSize;
    int32_t  width;
    int32_t  height;
    uint16_t planes;
    uint16_t bitDepth;
    uint32_t compression;
    uint32_t imageSize;
    int32_t  xPixelsPerMeter;
    int32_t  yPixelsPerMeter;
    uint32_t colorsUsed;
    uint32_t colorsImportant;
};
#pragma pack(pop)

struct Pixel {
    uint8_t blue;
    uint8_t green;
    uint8_t red;
};

struct Bitmap {
    BitmapHeader header;
    std::vector<Pixel> pixels;
};

void readBitmap(const char* filename, Bitmap& bitmap) {
    std::ifstream file(filename, std::ios::binary);

    if (!file.is_open()) {
        std::cerr << "Error opening file: " << filename << std::endl;
        return;
    }

    file.read(reinterpret_cast<char*>(&bitmap.header), sizeof(BitmapHeader));

    if (bitmap.header.signature != 0x4D42 || bitmap.header.bitDepth != 24 || bitmap.header.compression != 0) {
        std::cerr << "Invalid BMP file format or unsupported features." << std::endl;
        file.close();
        return;
    }

    uint32_t rowSize = (bitmap.header.width * 3 + 3) & ~3;
    bitmap.pixels.resize(bitmap.header.height * bitmap.header.width);

    for (int y = bitmap.header.height - 1; y >= 0; --y) {
        file.read(reinterpret_cast<char*>(bitmap.pixels.data() + y * bitmap.header.width), rowSize);
    }

    file.close();
}

void writeBitmap(const char* filename, const Bitmap& bitmap) {
    std::ofstream file(filename, std::ios::binary);

    if (!file.is_open()) {
        std::cerr << "Error opening file for writing: " << filename << std::endl;
        return;
    }

    file.write(reinterpret_cast<const char*>(&bitmap.header), sizeof(BitmapHeader));

    uint32_t rowSize = (bitmap.header.width * 3 + 3) & ~3;

    for (int y = bitmap.header.height - 1; y >= 0; --y) {
        file.write(reinterpret_cast<const char*>(bitmap.pixels.data() + y * bitmap.header.width), rowSize);
    }

    file.close();
}

__global__ void temp(Pixel *x, int width, int height) {
    int id_x = blockDim.x * blockIdx.x + threadIdx.x;
    int id_y = blockDim.y * blockIdx.y + threadIdx.y;
   

    if (id_x < width && id_y < height) {
        int id = id_y * width + id_x;

        if (blockIdx.x % 2 == 0 && blockIdx.y % 2 == 0) {
            x[id].red = 255 - x[id].red;
            x[id].blue = 255 - x[id].blue;
            x[id].green = 255 - x[id].green;
        }
    }
}

int main() {
    const char* filename = "input.bmp";
    Bitmap bitmap;

    readBitmap(filename, bitmap);
    std::cout << "Width: " << bitmap.header.width << std::endl;
    std::cout << "Height: " << bitmap.header.height << std::endl;

    Pixel *d_pixel;
    hipMalloc(&d_pixel, bitmap.header.height * bitmap.header.width * sizeof(Pixel));
    hipMemcpy(d_pixel, bitmap.pixels.data(), bitmap.header.height * bitmap.header.width * sizeof(Pixel), hipMemcpyHostToDevice);

    dim3 blockSize(32,32);  
    dim3 gridSize((bitmap.header.width + blockSize.x - 1) / blockSize.x, (bitmap.header.height + blockSize.y - 1) / blockSize.y);

    temp<<<gridSize, blockSize>>>(d_pixel, bitmap.header.width, bitmap.header.height);

    std::vector<Pixel> a(bitmap.header.height * bitmap.header.width);
    hipMemcpy(a.data(), d_pixel, bitmap.header.height * bitmap.header.width * sizeof(Pixel), hipMemcpyDeviceToHost);

    std::cout << "Modified Pixel - R: " << static_cast<int>(a[10].red) << " G: " << static_cast<int>(a[10].green) << " B: " << static_cast<int>(a[10].blue) << std::endl;

    Bitmap modifiedBitmap;
    modifiedBitmap.header = bitmap.header;
    modifiedBitmap.pixels = a;

    writeBitmap("output.bmp", modifiedBitmap);

    hipFree(d_pixel);

    return 0;
}

