#include "hip/hip_runtime.h"
%%cu
#include <algorithm>
#include <cassert>
#include <cstdlib>
#include <iostream>
#include <numeric>
#include <vector>

using std::accumulate;
using std::cout;
using std::generate;
using std::vector;

__global__ void temp(int *input, int *output)
{
    __shared__ int sum[256];

    int t_id = blockIdx.x * blockDim.x + threadIdx.x;

    sum[threadIdx.x] = input[t_id];
    __syncthreads();

    for (int s = 1; s < blockDim.x; s *= 2)
    {
        int index = 2 * s * threadIdx.x;
        if (index < blockDim.x)
        {
            sum[index] += sum[index + s];
        }
        __syncthreads();
    }

    if (threadIdx.x == 0)
        output[blockIdx.x] = sum[0];
}

void verify(const vector<int> &output)
{
   
    std::cout<<output[0]<<"\n";
    assert(output[0] == 65536);
}

int main()
{
    int N = 1<<16;
    size_t bytes = N * sizeof(int);

    vector<int> input(N);
    vector<int> output(N);

    for (int i = 0; i < N; i++)
    {
        input[i] = 1;
    }

    int *d_input, *d_output;

    hipMalloc(&d_input, bytes);
    hipMalloc(&d_output, bytes);
    hipMemcpy(d_input, input.data(), bytes, hipMemcpyHostToDevice);

    int threads = 256;
    int grids = N / threads;

    temp<<<grids, threads>>>(d_input, d_output);
    temp<<<1, threads>>>(d_output, d_output);

    hipMemcpy(output.data(), d_output, bytes, hipMemcpyDeviceToHost);

    verify(output);

    cout << "COMPLETED SUCCESSFULLY\n";

    return 0;
}

